
#include <hip/hip_runtime.h>
// CUDA_ART_getProj.cu
// many parts of the code might be improved (speed?? memory usage?) by using "index-pointers" ; always the same computations for different indiex-combinations ; don't know if the cuda compiler compresses this to safe memory for commands!

// in proj already raySum/rayWeight is kept!

__global__ void proj(const float * R, const int * volSize, const int * projSize, const float * vol, const float * recOrig, const float * origProj, float * proj) {
    // declare some useful varibles
    int count = 0;
    float det_pos_lab[3];
    float det_pos_sample[3];
    float ray_dir[3];
    short ray_sig[3];	
    float boundary_hit[3][2];
    float boundary_hit_pos[3][2][3];
    float current_k;
    int start_index[3];
    float start_pos[3];
    float t[3];
    float raySum = 0.0;
    //int internalProjSize[2];
    //internalProjSize[0] = (int) ((projSize[0] + 1 - gridTrace[0])/2.0);
    //internalProjSize[1] = (int) ((projSize[1] + 1 - gridTrace[1])/2.0);
    int noThreads = gridDim.x*gridDim.y*blockDim.x*blockDim.y*blockDim.z;
    int threadInd = (blockIdx.y*gridDim.x + blockIdx.x)*(blockDim.x*blockDim.y*blockDim.z) + blockDim.x*blockDim.y*threadIdx.z + blockDim.x*threadIdx.y + threadIdx.x; // just go through all indexed threads going to be started
    
    //while((count*noThreads+threadInd)<(internalProjSize[0]*internalProjSize[1])) {
    while((count*noThreads+threadInd)<(projSize[0]*projSize[1])) {
        // map the threads on the specified voxel indices (by gridTrace)
        int ind = threadInd+count*noThreads;
        //int indY = gridTrace[1] + 2*((int)(ind / internalProjSize[0]));
        //int indX = gridTrace[0] + 2*(ind - ((int)(ind / internalProjSize[0]))*internalProjSize[0]);
        int indY = ind / projSize[0];
        int indX = ind - indY*projSize[0];

        // adding 0.00001 to positions and directions in the case of even volSize but odd projection size leads to artifacts at the volume borders!
        det_pos_lab[0] = 0.0+0.00001;  // +0.00001 to avoid divisions by zero and other singularities of the raytracing algorithm --> better solution possible?!? (but shouldnot be a problem..)
        det_pos_lab[1] = indX+0.5-origProj[0]+0.00001;
        det_pos_lab[2] = indY+0.5-origProj[1]+0.00001;
        det_pos_sample[0] = R[0]*det_pos_lab[0] + R[3]*det_pos_lab[1] + R[6]*det_pos_lab[2] + recOrig[0] - volSize[0]/2.0;
        det_pos_sample[1] = R[1]*det_pos_lab[0] + R[4]*det_pos_lab[1] + R[7]*det_pos_lab[2] + recOrig[1] - volSize[1]/2.0;
        det_pos_sample[2] = R[2]*det_pos_lab[0] + R[5]*det_pos_lab[1] + R[8]*det_pos_lab[2] + recOrig[2] - volSize[2]/2.0;
        ray_dir[0] = R[0]; // might be improved by directly using R instead of an additional variable ray_dir // mustn't it be "ray_dir[i] = -R[i]" ?!?!?
        ray_dir[1] = R[1];
        ray_dir[2] = R[2];
        if (ray_dir[0]<0) ray_sig[0] = -1;
        if (ray_dir[0]>0) ray_sig[0] = 1;
        if (ray_dir[0]==0) {ray_sig[0] = 1; ray_dir[0] = 0.00001;} // is there a more elegant and safer (more general) way to avoid these indefinit expressions (divisions by zero)?!?
        if (ray_dir[1]<0) ray_sig[1] = -1;
        if (ray_dir[1]>0) ray_sig[1] = 1;
        if (ray_dir[1]==0) {ray_sig[1] = 1; ray_dir[1] = 0.00001;}
        if (ray_dir[2]<0) ray_sig[2] = -1;
        if (ray_dir[2]>0) ray_sig[2] = 1;
        if (ray_dir[2]==0) {ray_sig[2] = 1; ray_dir[2] = 0.00001;}

        // calculate intersections with the volume boundary
        // boundary_hit[0][0] = (volSize[0]/2.0-det_pos_sample[0]) / ray_dir[0];
        // boundary_hit[0][1] = (-volSize[0]/2.0-det_pos_sample[0]) / ray_dir[0];
        // boundary_hit[1][0] = (volSize[1]/2.0-det_pos_sample[1]) / ray_dir[1];
        // boundary_hit[1][1] = (-volSize[1]/2.0-det_pos_sample[1]) / ray_dir[1];
        // boundary_hit[2][0] = (volSize[2]/2.0-det_pos_sample[2]) / ray_dir[2];
        // boundary_hit[2][1] = (-volSize[2]/2.0-det_pos_sample[2]) / ray_dir[2];
        boundary_hit[0][0] = (volSize[0]/2.0-det_pos_sample[0]) / ray_dir[0];
        boundary_hit[0][1] = (-volSize[0]/2.0-det_pos_sample[0]) / ray_dir[0];
        boundary_hit[1][0] = (volSize[1]/2.0-det_pos_sample[1]) / ray_dir[1];
        boundary_hit[1][1] = (-volSize[1]/2.0-det_pos_sample[1]) / ray_dir[1];
        boundary_hit[2][0] = (volSize[2]/2.0-det_pos_sample[2]) / ray_dir[2];
        boundary_hit[2][1] = (-volSize[2]/2.0-det_pos_sample[2]) / ray_dir[2];

        // for the following optimization by using a loop possible? Or maybe put it all in a separate kernel and safe boundary matrix!
        boundary_hit_pos[0][0][0] = det_pos_sample[0] + ray_dir[0] * boundary_hit[0][0]; // might be optimized by only calculating the missing components and only until 2 possibilities are found!
        boundary_hit_pos[0][0][1] = det_pos_sample[1] + ray_dir[1] * boundary_hit[0][0];
        boundary_hit_pos[0][0][2] = det_pos_sample[2] + ray_dir[2] * boundary_hit[0][0];
        boundary_hit_pos[0][1][0] = det_pos_sample[0] + ray_dir[0] * boundary_hit[0][1]; // might be optimized by only calculating the missing components and only until 2 possibilities are found!
        boundary_hit_pos[0][1][1] = det_pos_sample[1] + ray_dir[1] * boundary_hit[0][1];
        boundary_hit_pos[0][1][2] = det_pos_sample[2] + ray_dir[2] * boundary_hit[0][1];
        boundary_hit_pos[1][0][0] = det_pos_sample[0] + ray_dir[0] * boundary_hit[1][0]; // might be optimized by only calculating the missing components and only until 2 possibilities are found!
        boundary_hit_pos[1][0][1] = det_pos_sample[1] + ray_dir[1] * boundary_hit[1][0];
        boundary_hit_pos[1][0][2] = det_pos_sample[2] + ray_dir[2] * boundary_hit[1][0];
        boundary_hit_pos[1][1][0] = det_pos_sample[0] + ray_dir[0] * boundary_hit[1][1]; // might be optimized by only calculating the missing components and only until 2 possibilities are found!
        boundary_hit_pos[1][1][1] = det_pos_sample[1] + ray_dir[1] * boundary_hit[1][1];
        boundary_hit_pos[1][1][2] = det_pos_sample[2] + ray_dir[2] * boundary_hit[1][1];
        boundary_hit_pos[2][0][0] = det_pos_sample[0] + ray_dir[0] * boundary_hit[2][0]; // might be optimized by only calculating the missing components and only until 2 possibilities are found!
        boundary_hit_pos[2][0][1] = det_pos_sample[1] + ray_dir[1] * boundary_hit[2][0];
        boundary_hit_pos[2][0][2] = det_pos_sample[2] + ray_dir[2] * boundary_hit[2][0];
        boundary_hit_pos[2][1][0] = det_pos_sample[0] + ray_dir[0] * boundary_hit[2][1]; // might be optimized by only calculating the missing components and only until 2 possibilities are found!
        boundary_hit_pos[2][1][1] = det_pos_sample[1] + ray_dir[1] * boundary_hit[2][1];
        boundary_hit_pos[2][1][2] = det_pos_sample[2] + ray_dir[2] * boundary_hit[2][1];

        current_k = 10000000000.0;
        short hit_count = 0;
        count++;

        // this part I do not like, yet: seems too complicated..
        if ( boundary_hit_pos[0][0][1]>-volSize[1]/2.0 && boundary_hit_pos[0][0][1]<volSize[1]/2.0 && boundary_hit_pos[0][0][2]>-volSize[2]/2.0 && boundary_hit_pos[0][0][2]<volSize[2]/2.0) {
            hit_count++;
            if (boundary_hit[0][0]<current_k) {
                current_k = boundary_hit[0][0];
                start_index[0] = volSize[0]-1;
                start_index[1] = (int) (boundary_hit_pos[0][0][1]+volSize[1]/2.0);
                start_index[2] = (int) (boundary_hit_pos[0][0][2]+volSize[2]/2.0);
                start_pos[0] = boundary_hit_pos[0][0][0];
                start_pos[1] = boundary_hit_pos[0][0][1];
                start_pos[2] = boundary_hit_pos[0][0][2];
                t[0] = ((float) ray_sig[0]) / ray_dir[0];
                // do it the easy way:
                if (ray_sig[1]>0) t[1] = (1.0 / ray_dir[1]) * (1.0 - ( start_pos[1] + volSize[1]/2.0 - (int) (start_pos[1] + volSize[1]/2.0) ) );
                else t[1] = (-1.0 / ray_dir[1]) * ( start_pos[1] + volSize[1]/2.0 - (int) (start_pos[1] + volSize[1]/2.0) );
                if (ray_sig[2]>0) t[2] = (1.0 / ray_dir[2]) * (1.0 - ( start_pos[2] + volSize[2]/2.0 - (int) (start_pos[2] + volSize[2]/2.0) ) );
                else t[2] = (-1.0 / ray_dir[2]) * ( start_pos[2] + volSize[2]/2.0 - (int) (start_pos[2] + volSize[2]/2.0) );
                }
            }
        if ( boundary_hit_pos[0][1][1]>-volSize[1]/2.0 && boundary_hit_pos[0][1][1]<volSize[1]/2.0 && boundary_hit_pos[0][1][2]>-volSize[2]/2.0 && boundary_hit_pos[0][1][2]<volSize[2]/2.0 ) {
            hit_count++;
            if (boundary_hit[0][1]<current_k) {
                current_k = boundary_hit[0][1];
                start_index[0] = 0;
                start_index[1] = (int) (boundary_hit_pos[0][1][1]+volSize[1]/2.0);
                start_index[2] = (int) (boundary_hit_pos[0][1][2]+volSize[2]/2.0);
                start_pos[0] = boundary_hit_pos[0][1][0];
                start_pos[1] = boundary_hit_pos[0][1][1];
                start_pos[2] = boundary_hit_pos[0][1][2];
                t[0] = ((float) ray_sig[0]) / ray_dir[0];
                if (ray_sig[1]>0) t[1] = (1.0 / ray_dir[1]) * (1.0 - ( start_pos[1] + volSize[1]/2.0 - (int) (start_pos[1] + volSize[1]/2.0) ) );
                else t[1] = (-1.0 / ray_dir[1]) * ( start_pos[1] + volSize[1]/2.0 - (int) (start_pos[1] + volSize[1]/2.0) );
                if (ray_sig[2]>0) t[2] = (1.0 / ray_dir[2]) * (1.0 - ( start_pos[2] + volSize[2]/2.0 - (int) (start_pos[2] + volSize[2]/2.0) ) );
                else t[2] = (-1.0 / ray_dir[2]) * ( start_pos[2] + volSize[2]/2.0 - (int) (start_pos[2] + volSize[2]/2.0) );
                }
            }
        if ( boundary_hit_pos[1][0][0]>-volSize[0]/2.0 && boundary_hit_pos[1][0][0]<volSize[0]/2.0 && boundary_hit_pos[1][0][2]>-volSize[2]/2.0 && boundary_hit_pos[1][0][2]<volSize[2]/2.0 ) {
            hit_count++;
            if (boundary_hit[1][0]<current_k) {
                current_k = boundary_hit[1][0];
                start_index[1] = volSize[1]-1;
                start_index[0] = (int) (boundary_hit_pos[1][0][0]+volSize[0]/2.0);
                start_index[2] = (int) (boundary_hit_pos[1][0][2]+volSize[2]/2.0);
                start_pos[0] = boundary_hit_pos[1][0][0];
                start_pos[1] = boundary_hit_pos[1][0][1];
                start_pos[2] = boundary_hit_pos[1][0][2];
                t[1] = ((float) ray_sig[1]) / ray_dir[1];
                if (ray_sig[0]>0) t[0] = (1.0 / ray_dir[0]) * (1.0 - ( start_pos[0] + volSize[0]/2.0 - (int) (start_pos[0] + volSize[0]/2.0) ) );
                else t[0] = (-1.0 / ray_dir[0]) * ( start_pos[0] + volSize[0]/2.0 - (int) (start_pos[0] + volSize[0]/2.0) );
                if (ray_sig[2]>0) t[2] = (1.0 / ray_dir[2]) * (1.0 - ( start_pos[2] + volSize[2]/2.0 - (int) (start_pos[2] + volSize[2]/2.0) ) );
                else t[2] = (-1.0 / ray_dir[2]) * ( start_pos[2] + volSize[2]/2.0 - (int) (start_pos[2] + volSize[2]/2.0) );
                }
            }
        if ( boundary_hit_pos[1][1][0]>-volSize[0]/2.0 && boundary_hit_pos[1][1][0]<volSize[0]/2.0 && boundary_hit_pos[1][1][2]>-volSize[2]/2.0 && boundary_hit_pos[1][1][2]<volSize[2]/2.0 ) {
            hit_count++;
            if (boundary_hit[1][1]<current_k) {
                current_k = boundary_hit[1][1];
                start_index[1] = 0;
                start_index[0] = (int) (boundary_hit_pos[1][1][0]+volSize[0]/2.0);
                start_index[2] = (int) (boundary_hit_pos[1][1][2]+volSize[2]/2.0);
                start_pos[0] = boundary_hit_pos[1][1][0];
                start_pos[1] = boundary_hit_pos[1][1][1];
                start_pos[2] = boundary_hit_pos[1][1][2];
                t[1] = ((float) ray_sig[1]) / ray_dir[1];
                if (ray_sig[0]>0) t[0] = (1.0 / ray_dir[0]) * (1.0 - ( start_pos[0] + volSize[0]/2.0 - (int) (start_pos[0] + volSize[0]/2.0) ) );
                else t[0] = (-1.0 / ray_dir[0]) * ( start_pos[0] + volSize[0]/2.0 - (int) (start_pos[0] + volSize[0]/2.0) );
                if (ray_sig[2]>0) t[2] = (1.0 / ray_dir[2]) * (1.0 - ( start_pos[2] + volSize[2]/2.0 - (int) (start_pos[2] + volSize[2]/2.0) ) );
                else t[2] = (-1.0 / ray_dir[2]) * ( start_pos[2] + volSize[2]/2.0 - (int) (start_pos[2] + volSize[2]/2.0) );
                }
            }
        if ( boundary_hit_pos[2][0][0]>-volSize[0]/2.0 && boundary_hit_pos[2][0][0]<volSize[0]/2.0 && boundary_hit_pos[2][0][1]>-volSize[1]/2.0 && boundary_hit_pos[2][0][1]<volSize[1]/2.0 ) {
            hit_count++;
            if (boundary_hit[2][0]<current_k) {
                current_k = boundary_hit[2][0];
                start_index[2] = volSize[2]-1;
                start_index[1] = (int) (boundary_hit_pos[2][0][1]+volSize[1]/2.0);
                start_index[0] = (int) (boundary_hit_pos[2][0][0]+volSize[0]/2.0);
                start_pos[0] = boundary_hit_pos[2][0][0];
                start_pos[1] = boundary_hit_pos[2][0][1];
                start_pos[2] = boundary_hit_pos[2][0][2];
                t[2] = ((float) ray_sig[2]) / ray_dir[2];
                if (ray_sig[0]>0) t[0] = (1.0 / ray_dir[0]) * (1.0 - ( start_pos[0] + volSize[0]/2.0 - (int) (start_pos[0] + volSize[0]/2.0) ) );
                else t[0] = (-1.0 / ray_dir[0]) * ( start_pos[0] + volSize[0]/2.0 - (int) (start_pos[0] + volSize[0]/2.0) );
                if (ray_sig[1]>0) t[1] = (1.0 / ray_dir[1]) * (1.0 - ( start_pos[1] + volSize[1]/2.0 - (int) (start_pos[1] + volSize[1]/2.0) ) );
                else t[1] = (-1.0 / ray_dir[1]) * ( start_pos[1] + volSize[1]/2.0 - (int) (start_pos[1] + volSize[1]/2.0) );
                }
            }
        if ( boundary_hit_pos[2][1][0]>-volSize[0]/2.0 && boundary_hit_pos[2][1][0]<volSize[0]/2.0 && boundary_hit_pos[2][1][1]>-volSize[1]/2.0 && boundary_hit_pos[2][1][1]<volSize[1]/2.0 ) {
            hit_count++;
            if (boundary_hit[2][1]<current_k) {
                current_k = boundary_hit[2][1];
                start_index[2] = 0;
                start_index[1] = (int) (boundary_hit_pos[2][1][1]+volSize[1]/2.0);
                start_index[0] = (int) (boundary_hit_pos[2][1][1]+volSize[2]/2.0);
                start_pos[0] = boundary_hit_pos[2][1][0];
                start_pos[1] = boundary_hit_pos[2][1][1];
                start_pos[2] = boundary_hit_pos[2][1][2];
                t[2] = ((float) ray_sig[2]) / ray_dir[2];
                if (ray_sig[0]>0) t[0] = (1.0 / ray_dir[0]) * (1.0 - ( start_pos[0] + volSize[0]/2.0 - (int) (start_pos[0] + volSize[0]/2.0) ) );
                else t[0] = (-1.0 / ray_dir[0]) * ( start_pos[0] + volSize[0]/2.0 - (int) (start_pos[0] + volSize[0]/2.0) );
                if (ray_sig[1]>0) t[1] = (1.0 / ray_dir[1]) * (1.0 - ( start_pos[1] + volSize[1]/2.0 - (int) (start_pos[1] + volSize[1]/2.0) ) );
                else t[1] = (-1.0 / ray_dir[1]) * ( start_pos[1] + volSize[1]/2.0 - (int) (start_pos[1] + volSize[1]/2.0) );
                }
            }
        if ( hit_count < 2 ) continue;

        float ray_length = 0.0;
        int X = start_index[0];
        int Y = start_index[1];
        int Z = start_index[2];

        raySum = 0.0;

        // the voxel ray tracing looks quite nice and efficient:
        while (X>=0 && X<volSize[0] && Y>=0 && Y<volSize[1] && Z>=0 && Z<volSize[2]) {
            if (t[0]<t[1]) {
                if (t[2]<t[0]) {
                    // 2=min:
                    t[0] -= t[2]; t[1] -= t[2];
                    raySum += t[2]*vol[Z*volSize[0]*volSize[1]+Y*volSize[0]+X];
                    //proj[indY*projSize[0]+indX] += t[2]*vol[Z*volSize[0]*volSize[1]+Y*volSize[0]+X];
                    //vol[Z*volSize[0]*volSize[1]+Y*volSize[0]+X] += t[2]*proj[indY*projSize[0]+indX];
                    ray_length += t[2];
                    t[2] = ((float) ray_sig[2]) / ray_dir[2];
                    Z += ray_sig[2];
                } else {
                    // 0=min:
                    t[1] -= t[0]; t[2] -= t[0];
                    raySum += t[0]*vol[Z*volSize[0]*volSize[1]+Y*volSize[0]+X];
                    //proj[indY*projSize[0]+indX] += t[0]*vol[Z*volSize[0]*volSize[1]+Y*volSize[0]+X];
                    //vol[Z*volSize[0]*volSize[1]+Y*volSize[0]+X] += t[0]*proj[indY*projSize[0]+indX];
                    ray_length += t[0];
                    t[0] = ((float) ray_sig[0]) / ray_dir[0];
                    X += ray_sig[0];
                }
            } else {
                if (t[2]<t[1]) {
                    // 2=min:
                    t[0] -= t[2]; t[1] -= t[2];
                    raySum += t[2]*vol[Z*volSize[0]*volSize[1]+Y*volSize[0]+X];
                    //proj[indY*projSize[0]+indX] += t[2]*vol[Z*volSize[0]*volSize[1]+Y*volSize[0]+X];
                    //vol[Z*volSize[0]*volSize[1]+Y*volSize[0]+X] += t[2]*proj[indY*projSize[0]+indX];
                    ray_length += t[2];
                    t[2] = ((float) ray_sig[2]) / ray_dir[2];
                    Z += ray_sig[2];
                } else {
                    // 1=min:
                    t[0] -= t[1]; t[2] -= t[1];
                    raySum += t[1]*vol[Z*volSize[0]*volSize[1]+Y*volSize[0]+X];
                    //proj[indY*projSize[0]+indX] += t[1]*vol[Z*volSize[0]*volSize[1]+Y*volSize[0]+X];
                    //vol[Z*volSize[0]*volSize[1]+Y*volSize[0]+X] += t[1]*proj[indY*projSize[0]+indX];
                    ray_length += t[1];
                    t[1] = ((float) ray_sig[1]) / ray_dir[1];
                    Y += ray_sig[1];
                }
            }
        }
        proj[indY*projSize[0]+indX] -= raySum;
        proj[indY*projSize[0]+indX] /= ray_length; 
    }
}